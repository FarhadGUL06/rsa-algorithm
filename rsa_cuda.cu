#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <iostream>

using namespace std;

uint64_t public_key;
uint64_t private_key;
uint64_t n;

int num_blocks = 65535;
const int num_threads = 1024;

const size_t size_of_ciur = 500;
const uint64_t size_array = 10000005;

// All headers for functions from bellow

uint64_t gcd(uint64_t a, uint64_t h);
size_t primefiller(uint64_t *primes);
uint64_t pickrandomprime(uint64_t *primes, size_t no_primes, int64_t *pos);
void setkeys(uint64_t *primes, size_t no_primes);
uint64_t encrypt(uint8_t message);
uint8_t decrypt(uint64_t encrpyted_text);
uint64_t *stringToNumbersArray(char *str);
char *numberArrayToString(uint64_t *numbers, size_t size);

/**
    Functie care calculeaza cel mai mare divizor comun
    pentru 2 numere intregi

    @param a primul paremetrul -> uint64_t
    @param h al doilea parametru -> uint64_t
    @return h – cel mai mare divizor comun -> uint64_t
*/
uint64_t gcd(uint64_t a, uint64_t h) {
    uint64_t temp;
    while (1) {
        temp = a % h;
        if (temp == 0) return h;
        a = h;
        h = temp;
    }
}

/**
    Functia care construieste ciurul lui Eratosthenes

    @param primes care va referentia un array populat cu numere prime ->
   uint64_t
    @return size_prime – marimea array-ului de numere prime –> size_t
*/
size_t primefiller(uint64_t *primes) {
    size_t size_prime = 0;
    uint8_t *ciur = (uint8_t *)malloc(size_of_ciur * sizeof(uint8_t) + 1);
    memset(ciur, 1, size_of_ciur * sizeof(uint8_t) + 1);

    ciur[0] = false;
    ciur[1] = false;
    for (size_t i = 2; i < size_of_ciur; i++) {
        for (size_t j = i * 2; j < size_of_ciur; j += i) {
            ciur[j] = false;
        }
    }
    for (size_t i = 0; i < size_of_ciur; i++) {
        if (ciur[i]) {
            primes[size_prime] = i;
            ++size_prime;
        }
    }
    free(ciur);
    return size_prime;
}

/**
    Functie care alege un numar random prim

    @param primes array-ul de numere prime -> uint64_t
    @param no_primes numarul de numere prime -> size_t
    @param pos retine pozitia anterioara pentru a pastra diferenta intre prime1
   si prime2 -> uint64_t
    @return primes[k] – numarul prim de la pozitia k -> uint64_t
*/
uint64_t pickrandomprime(uint64_t *primes, size_t no_primes, uint64_t *pos) {
    uint64_t k = rand() % no_primes;
    while (k == *pos) {
        k = rand() % no_primes;
    }
    *pos = k;
    return primes[k];
}

/**
    Functie care construieste cheia publica si
    cheia privata pornind de la 2 numere prime
    generate folosind functia `pickrandomprime`

    @param primes array-ul de numere prime -> uint64_t
    @param no_primes numarul de numere prime -> size_t
*/
void setkeys(uint64_t *primes, size_t no_primes) {
    uint64_t pos = 0;
    uint64_t prime1 = pickrandomprime(primes, no_primes, &pos);  // 17291
    uint64_t prime2 = pickrandomprime(primes, no_primes, &pos);  // 64817

    n = prime1 * prime2;

    uint64_t phi = (prime1 - 1) * (prime2 - 1);

    uint64_t e = 2;

    while (1) {
        if (gcd(e, phi) == 1) {
            break;
        }
        e++;
    }

    public_key = e;
    int d = 2;

    while (1) {
        if ((d * e) % phi == 1) {
            break;
        }
        d++;
    }

    private_key = d;

    cout << "Public key: " << public_key << endl;
    cout << "Private key: " << private_key << endl;
}

/**
    Functie de encriptare a unui caracter

    @param message caracterul ce trebuie encriptat -> uint8_t
    @return encrpyted_text – caracterul encriptat -> uint64_t
*/
uint64_t encrypt(uint8_t message) {
    uint64_t e = public_key;
    uint64_t encrpyted_text = 1;
    while (e > 0) {
        encrpyted_text *= message;
        encrpyted_text %= n;
        --e;
    }
    return encrpyted_text;
}

/**
    Functie de decriptare a unui numar

    @param encrypted_text caracterul ce trebuie decriptat -> uint64_t
    @return decrypted – caracterul decriptat -> uint8_t
*/
uint8_t decrypt(uint64_t encrpyted_text) {
    uint64_t d = private_key;
    uint64_t decrypted = 1;

    while (d > 0) {
        decrypted *= encrpyted_text;
        decrypted %= n;
        --d;
    }
    return (uint8_t)decrypted;
}

__global__ void parallel_decrypt(char *d_str, uint64_t *d_numbers, size_t size,
                                 uint64_t *d_private_key, uint64_t *d_n) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index > size) {
        // Indice mai mare decat lungimea vectorului de inserat
        return;
    }
    // d_str[index] = decrypt(d_numbers[index]);
    uint64_t encrpyted_text = d_numbers[index];

    uint64_t d = *d_private_key;
    // printf("Cheie priv: %ld\n", d);
    uint64_t decrypted = 1;
    while (d > 0) {
        decrypted *= encrpyted_text;
        decrypted %= *d_n;
        --d;
    }
    d_str[index] = decrypted;
}

__global__ void parallel_encrypt(uint64_t *d_numbers, char *d_str, size_t size,
                                 uint64_t *d_public_key, uint64_t *d_n) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index > size) {
        // Indice mai mare decat lungimea vectorului de inserat
        return;
    }
    uint64_t e = *d_public_key;
    uint64_t encrpyted_text = 1;
    while (e > 0) {
        encrpyted_text *= d_str[index];
        encrpyted_text %= *d_n;
        --e;
    }
    d_numbers[index] = encrpyted_text;
}

/**
   Convertirea unui string la char array

   @param str stringul de convertit (trimis ca char*)
   @return numbers – array-ul de numere
*/
uint64_t *stringToNumbersArray(char **h_str) {
    char *d_str;
    uint64_t size = strlen(*h_str) + 1;
    hipMalloc((void **)&d_str, size * sizeof(char));
    hipMemset(d_str, 0, size * sizeof(char));
    hipMemcpy(d_str, *h_str, size * sizeof(char), hipMemcpyHostToDevice);

    uint64_t *d_numbers;
    hipMalloc((void **)&d_numbers, size * sizeof(uint64_t));
    hipMemset(d_numbers, 0, size * sizeof(uint64_t));

    /*for (size_t i = 0; i < strlen(*h_str); ++i) {
        numbers[i] = encrypt((uint64_t)(*h_str[i]));
    }*/
    uint64_t *d_public_key;
    uint64_t *d_n;
    hipMalloc((void **)&d_public_key, sizeof(uint64_t));
    hipMalloc((void **)&d_n, sizeof(uint64_t));
    hipMemcpy(d_public_key, &public_key, sizeof(uint64_t),
               hipMemcpyHostToDevice);
    hipMemcpy(d_n, &n, sizeof(uint64_t), hipMemcpyHostToDevice);

    // Execute on gpu encrypt
    parallel_encrypt<<<num_blocks, num_threads>>>(
        d_numbers, d_str, strlen(*h_str), d_public_key, d_n);
    hipDeviceSynchronize();

    uint64_t *h_numbers = (uint64_t *)malloc(size * sizeof(uint64_t));
    memset(h_numbers, 0, size * sizeof(uint64_t));
    hipMemcpy(h_numbers, d_numbers, size * sizeof(uint64_t),
               hipMemcpyDeviceToHost);
    hipFree(d_numbers);
    hipFree(d_str);
    return h_numbers;
}

/**
   Convertirea unui char array la string

   @param numbers array-ul de numere -> uint64_t
   @param size marimea array-ului -> size_t
   @return str – textul decriptat -> char*
*/
char *numberArrayToString(uint64_t **h_numbers, size_t size) {
    // Copy numbers to GPU
    uint64_t *d_numbers;
    hipMalloc((void **)&d_numbers, size * sizeof(uint64_t));
    hipMemset(d_numbers, 0, size * sizeof(uint64_t));
    hipMemcpy(d_numbers, *h_numbers, size * sizeof(uint64_t),
               hipMemcpyHostToDevice);

    // Initialise d_str
    char *d_str;
    hipMalloc((void **)&d_str, size * sizeof(char));
    hipMemset(d_str, 0, size * sizeof(char));

    /*for (size_t i = 0; i < size; ++i) {
        h_str[i] = decrypt(*h_numbers[i]);
    }*/

    // Copy data for decrypt
    uint64_t *d_private_key;
    uint64_t *d_n;
    hipMalloc((void **)&d_private_key, sizeof(uint64_t));
    hipMalloc((void **)&d_n, sizeof(uint64_t));

    hipMemcpy(d_private_key, &private_key, sizeof(uint64_t),
               hipMemcpyHostToDevice);
    hipMemcpy(d_n, &n, sizeof(uint64_t), hipMemcpyHostToDevice);

    // Execute on gpu decrypt
    parallel_decrypt<<<num_blocks, num_threads>>>(d_str, d_numbers, size,
                                                  d_private_key, d_n);
    hipDeviceSynchronize();

    // Copy str from device to host
    char *h_str = (char *)malloc(size * sizeof(char));
    memset(h_str, 0, size * sizeof(char));
    hipMemcpy(h_str, d_str, size * sizeof(char), hipMemcpyDeviceToHost);
    hipFree(d_str);
    hipFree(d_numbers);
    return h_str;
}

int main() {
    fflush(stdin);
    fflush(stdout);
    
    srand(time(NULL));

    uint64_t *primes = (uint64_t *)malloc(size_of_ciur * sizeof(uint64_t));
    memset(primes, 0, size_of_ciur * sizeof(uint64_t));

    size_t no_primes = primefiller(primes);

    setkeys(primes, no_primes);

    char *message = (char *)malloc(size_array * sizeof(char));
    char *p = fgets(message, size_array, stdin);

    if (p == NULL) {
        return -1;
    }

    int sizeOfMessage = strlen(message) + 1;

    uint64_t *numbers = stringToNumbersArray(&message);
    printf("Criptat: ");
    for (int i = 0; i < sizeOfMessage; i++) {
        printf("%lu ", numbers[i]);
    }
    printf("\n");

    char *str = numberArrayToString(&numbers, sizeOfMessage);

    printf("Decriptat: %s\n", str);

    free(primes);
    free(numbers);
    free(str);

    return 0;
}
